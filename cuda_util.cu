#include "cuda_util.cuh"

/**
 * @brief Wrapper around hipMallocManaged available to non-Cuda code.
 */
void cuda_malloc_managed(void*& ptr, size_t bytes) {
    cudachk(hipMallocManaged(&ptr, bytes));
    hipDeviceSynchronize();
}

/**
 * @brief Wrapper around hipFree available to non-Cuda code.
 */
void cuda_free(void* ptr) {
    hipDeviceSynchronize();
    cudachk(hipFree(ptr));
}