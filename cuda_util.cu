#include "cuda_util.cuh"

/**
 * @brief Wrapper around hipMallocManaged available to non-Cuda code.
 */
void cuda_malloc_managed(void*& ptr, size_t bytes) {
    cudachk(hipMallocManaged(&ptr, bytes));
}

/**
 * @brief Wrapper around hipFree available to non-Cuda code.
 */
void cuda_free(void* ptr) { cudachk(hipFree(ptr)); }
