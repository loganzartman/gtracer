#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <cuda_gl_interop.h>
#include <cassert>
#include <iostream>
#include "cuda_render.cuh"

void cuda_init(GLuint texture_id, GLuint buffer_id) {
    // register GL buffer and texture as CUDA resources
    hipGraphicsGLRegisterBuffer(&cuda_buffer, buffer_id,
                                 hipGraphicsRegisterFlagsNone);
    hipGraphicsGLRegisterImage(&cuda_texture, texture_id, GL_TEXTURE_2D,
                                hipGraphicsRegisterFlagsNone);

    // create CUDA stream
}

void cuda_render(GLuint buffer_id, size_t w, size_t h, const Mat4f &camera,
                 std::vector<Geometry *> geom, unsigned iteration) {
    using namespace std;

    const size_t size_pixels = w * h;
    float *mem_ptr;
    hipArray *array_ptr;

    // map resources
    hipStreamCreate(&hip_stream);
    hipGraphicsMapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsMapResources(1, &cuda_texture, hip_stream);

    size_t size_mapped;
    hipGraphicsSubResourceGetMappedArray(&array_ptr, cuda_texture, 0, 0);
    hipGraphicsResourceGetMappedPointer((void **)&mem_ptr, &size_mapped,
                                         cuda_buffer);
    assert(size_mapped == size_pixels * 4 * sizeof(float));  // RGBA32F

    // run kernel
    CUDAKernelArgs args = {w, h, iteration, mem_ptr};
    const int num_blocks = (size_pixels + BLOCK_SIZE - 1) / BLOCK_SIZE;
    cuda_render_kernel<<<num_blocks, BLOCK_SIZE>>>(args);

    // unmap resources
    hipGraphicsUnmapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsUnmapResources(1, &cuda_texture, hip_stream);
    hipStreamDestroy(hip_stream);
}

void cuda_destroy() {}

__global__ void cuda_render_kernel(CUDAKernelArgs args) {
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;

    const size_t len = args.w * args.h;
    for (size_t i = index; i < len; i += stride) {
        const size_t idx = i * 4;
        const size_t x = i % args.w;
        const size_t y = i / args.w;
        float fx = (float)x / args.w, fy = (float)y / args.h;
        args.pixels[idx + 0] = sin(fx + args.iteration * 0.1f);
        args.pixels[idx + 1] = cos(fy + args.iteration * 0.02f);
        args.pixels[idx + 2] = 0.f;
        args.pixels[idx + 3] = 1.f;
    }
}