#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <cuda_gl_interop.h>
#include <cassert>
#include <iostream>
#include "cuda_render.cuh"

void cuda_init(GLuint texture_id, GLuint buffer_id) {
    // register GL buffer and texture as CUDA resources
    hipGraphicsGLRegisterBuffer(&cuda_buffer, buffer_id,
                                 hipGraphicsRegisterFlagsNone);
    hipGraphicsGLRegisterImage(&cuda_texture, texture_id, GL_TEXTURE_2D,
                                hipGraphicsRegisterFlagsNone);

    // create CUDA stream
}

void cuda_render(GLuint buffer_id, size_t w, size_t h, const Mat4f &camera,
                 std::vector<Geometry *> geom, unsigned iteration) {
    using namespace std;

    const size_t size_pixels = w * h;
    float *mem_ptr;
    hipArray *array_ptr;

    // map resources
    hipStreamCreate(&hip_stream);
    hipGraphicsMapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsMapResources(1, &cuda_texture, hip_stream);

    size_t size_mapped;
    hipGraphicsSubResourceGetMappedArray(&array_ptr, cuda_texture, 0, 0);
    hipGraphicsResourceGetMappedPointer((void **)&mem_ptr, &size_mapped,
                                         cuda_buffer);
    assert(size_mapped == size_pixels * 4 * sizeof(float));  // RGBA32F

    // run kernel
    const int num_blocks = (size_pixels + BLOCK_SIZE - 1) / BLOCK_SIZE;
    cuda_render_test_kernel<<<num_blocks, BLOCK_SIZE>>>(w, h, mem_ptr);

    // unmap resources
    hipGraphicsUnmapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsUnmapResources(1, &cuda_texture, hip_stream);
    hipStreamDestroy(hip_stream);
}

void cuda_destroy() {}

__global__ void cuda_render_test_kernel(size_t w, size_t h, float *mem_ptr) {
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < w * h; i += stride) {
        const size_t idx = i * 4;
        const size_t x = i % w;
        const size_t y = i / w;
        mem_ptr[idx + 0] = (float)x / w;
        mem_ptr[idx + 1] = (float)y / w;
        mem_ptr[idx + 2] = 1.f - fabs(((float)y / h) - 0.5f) * 2;
        mem_ptr[idx + 3] = 1.f;
    }
}