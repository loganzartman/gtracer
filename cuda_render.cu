#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <cuda_gl_interop.h>
#include <algorithm>
#include <cassert>
#include <cmath>
#include <iostream>
#include <memory>
#include <stdexcept>
#include <vector>

#include "Box.hh"
#include "Geometry.hh"
#include "Mat.hh"
#include "Material.hh"
#include "Sphere.hh"
#include "Tri.hh"
#include "UniformGrid.hh"
#include "Vec3.hh"
#include "cuda_render.cuh"
#include "raytracing.hh"
#include "transform.hh"
#include "util.hh"

/**
 * @brief Initializes CUDA resources.
 * @detail Called once upon program start. Registers GL texture and buffer
 * for CUDA/GL interop; creates stream and maps buffer and texture to stream.
 *
 * @param texture_id ID of the GL texture
 * @param buffer_id  ID of the GL buffer
 */
void cuda_init(GLuint texture_id, GLuint buffer_id) {
    // register GL buffer and texture as CUDA resources
    hipGraphicsGLRegisterBuffer(&cuda_buffer, buffer_id,
                                 hipGraphicsRegisterFlagsNone);
    hipGraphicsGLRegisterImage(&cuda_texture, texture_id, GL_TEXTURE_2D,
                                hipGraphicsRegisterFlagsNone);

    // create CUDA stream
    hipStreamCreate(&hip_stream);

    // map resources
    hipGraphicsMapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsMapResources(1, &cuda_texture, hip_stream);
}

void cuda_render(GLuint buffer_id, size_t w, size_t h, const Mat4f &camera,
                 Geometry *geom, size_t geom_len, unsigned iteration, bool accel) {
    using namespace std;

    const size_t size_pixels = w * h;
    float *mem_ptr;
    hipArray *array_ptr;

    size_t size_mapped;
    hipGraphicsSubResourceGetMappedArray(&array_ptr, cuda_texture, 0, 0);
    hipGraphicsResourceGetMappedPointer((void **)&mem_ptr, &size_mapped,
                                         cuda_buffer);
    // assert(size_mapped == size_pixels * 4 * sizeof(float));  // RGBA32F

    // construct uniform grid
    AABB bounds = geometry_bounds(geom, geom + geom_len);
    Int3 res = UniformGrid::resolution(bounds, geom_len);
    size_t n_data = UniformGrid::data_size(res);
    size_t n_pairs =
        UniformGrid::count_pairs(res, bounds, geom, geom + geom_len);
    ugrid_data_t *grid_data;
    ugrid_pair_t *grid_pairs;
    hipMallocManaged(&grid_data, n_data * sizeof(ugrid_data_t));
    hipMallocManaged(&grid_pairs, n_pairs * sizeof(ugrid_pair_t));
    hipDeviceSynchronize();
    UniformGrid grid(res, bounds, grid_data, grid_pairs, n_pairs, geom,
                     geom + geom_len);

    // run kernel
    CUDAKernelArgs args = {w, h, camera, bounds, grid, accel, iteration, mem_ptr};
    const int num_blocks = (size_pixels + BLOCK_SIZE - 1) / BLOCK_SIZE;
    cuda_render_kernel<<<num_blocks, BLOCK_SIZE>>>(args);

    reinhard(mem_ptr, w, h);

    hipDeviceSynchronize();
    hipFree(grid_data);
    hipFree(grid_pairs);
}

/**
 * @brief Destroy resource
 *
 */
void cuda_destroy() {
    // unmap resources
    hipGraphicsUnmapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsUnmapResources(1, &cuda_texture, hip_stream);
    hipStreamDestroy(hip_stream);
}

/**
 * @brief Path tracing kernel
 * @param args current state
 */
__global__ void cuda_render_kernel(CUDAKernelArgs args) {
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;

    float inv_w = 1 / float(args.w);
    float inv_h = 1 / float(args.h);
    float fov = 30;
    float aspect_ratio = float(args.w) / float(args.h);
    float angle = tan(0.5 * M_PI * fov / 180.0);

    Mat4f dir_camera = transform_clear_translate(args.camera);
    Float3 origin = args.camera * Float3();

    const size_t len = args.w * args.h;
    for (size_t i = index; i < len; i += stride) {
        const size_t idx = i * 4;
        const size_t x = i % args.w;
        const size_t y = i / args.w;

        Float3 color;
        for (size_t i = 0; i < PRIMARY_RAYS; ++i) {
            float v_x = (2 * ((x + util::randf(0, 1)) * inv_w) - 1) * angle *
                        aspect_ratio;
            float v_y = (1 - 2 * ((y + util::randf(0, 1)) * inv_h)) * angle;
            Float3 ray_dir = dir_camera * Float3(v_x, v_y, -1);
            ray_dir.normalize();

            color += raytracing::trace(origin, ray_dir, args.bounds, args.grid, args.accel, 8);
        }
        color *= 1.f / PRIMARY_RAYS;

        // compute all-time average color
        Float3 dst = Float3(args.pixels[idx], args.pixels[idx + 1],
                            args.pixels[idx + 2]);
        float f = 1;
        if (args.iteration > 0)
            f = 1.f / args.iteration;
        Float3 blended = color * f + dst * (1 - f);

        // write color
        args.pixels[idx] = blended.x;
        args.pixels[idx + 1] = blended.y;
        args.pixels[idx + 2] = blended.z;
        args.pixels[idx + 3] = 1;  // alpha
    }
}
