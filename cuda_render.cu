#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <cuda_gl_interop.h>
#include <cassert>
#include <iostream>
#include <vector>
#include "cuda_render.cuh"
#include "Geometry.hh"
#include "Sphere.hh"
#include "Tri.hh"
#include "Box.hh"

/**
 * @brief Initializes CUDA resources.
 * @detail Called once upon program start. Registers GL texture and buffer
 * for CUDA/GL interop; creates stream and maps buffer and texture to stream.
 * 
 * @param texture_id ID of the GL texture
 * @param buffer_id  ID of the GL buffer
 */
void cuda_init(GLuint texture_id, GLuint buffer_id, const std::vector<Geometry*>& geom) {
    // register GL buffer and texture as CUDA resources
    hipGraphicsGLRegisterBuffer(&cuda_buffer, buffer_id,
                                 hipGraphicsRegisterFlagsNone);
    hipGraphicsGLRegisterImage(&cuda_texture, texture_id, GL_TEXTURE_2D,
                                hipGraphicsRegisterFlagsNone);

    // create CUDA stream
    hipStreamCreate(&hip_stream);

    // map resources
    hipGraphicsMapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsMapResources(1, &cuda_texture, hip_stream);
}

void cuda_render(GLuint buffer_id, size_t w, size_t h, const Mat4f &camera,
                 std::vector<Geometry *> geom, unsigned iteration) {
    using namespace std;

    const size_t size_pixels = w * h;
    float *mem_ptr;
    hipArray *array_ptr;

    size_t size_mapped;
    hipGraphicsSubResourceGetMappedArray(&array_ptr, cuda_texture, 0, 0);
    hipGraphicsResourceGetMappedPointer((void **)&mem_ptr, &size_mapped,
                                         cuda_buffer);
    //assert(size_mapped == size_pixels * 4 * sizeof(float));  // RGBA32F

    Geometry **dev_geom;
    hipMallocManaged(&dev_geom, geom.size());

    cuda_update_geometry(geom, dev_geom);

    // run kernel
    CUDAKernelArgs args = {w, h, iteration, mem_ptr, dev_geom};
    const int num_blocks = (size_pixels + BLOCK_SIZE - 1) / BLOCK_SIZE;
    cuda_render_kernel<<<num_blocks, BLOCK_SIZE>>>(args);
}

/**
 * @brief Copies a new set of Geometry to the GPU
 * @detail allocates memory and uses kernel to construct Geometries on GPU
 * 
 * @param geom the vector of geometry to copy
 */
void cuda_update_geometry(const std::vector<Geometry*>& geom, Geometry** dev_geom) {
    Geometry *x;
    for (size_t i = 0; i < geom.size(); ++i) {
        hipMallocManaged(&x, sizeof(decltype(*geom[i])));
        dev_geom[i] = x;
    }
}

/**
 * @brief Destroy resource
 * 
 */
void cuda_destroy() {
    // unmap resources
    hipGraphicsUnmapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsUnmapResources(1, &cuda_texture, hip_stream);
    hipStreamDestroy(hip_stream);
}

/**
 * @brief Path tracing kernel
 * @param args current state 
 */
__global__ void cuda_render_kernel(CUDAKernelArgs args) {
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;

    const size_t len = args.w * args.h;
    for (size_t i = index; i < len; i += stride) {
        const size_t idx = i * 4;
        const size_t x = i % args.w;
        const size_t y = i / args.w;
        float fx = (float)x / args.w, fy = (float)y / args.h;
        args.pixels[idx + 0] = sin(fx + args.iteration * 0.1f);
        args.pixels[idx + 1] = cos(fy + args.iteration * 0.02f);
        args.pixels[idx + 2] = 0.f;
        args.pixels[idx + 3] = 1.f;
    }
}
