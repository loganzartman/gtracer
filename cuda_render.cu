#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <cuda_gl_interop.h>
#include <cassert>
#include <iostream>
#include "cuda_render.cuh"

void cuda_init(GLuint texture_id, GLuint buffer_id) {
    // register GL buffer and texture as CUDA resources
    hipGraphicsGLRegisterBuffer(&cuda_buffer, buffer_id,
                                 hipGraphicsRegisterFlagsNone);
    hipGraphicsGLRegisterImage(&cuda_texture, texture_id, GL_TEXTURE_2D,
                                hipGraphicsRegisterFlagsNone);

    // create CUDA stream
}

void cuda_render(GLuint buffer_id, size_t w, size_t h, const Mat4f &camera,
                 std::vector<Geometry *> geom, unsigned iteration) {
    using namespace std;

    const size_t size_pixels = w * h;
    float *mem_ptr;
    hipArray *array_ptr;

    // map resources
    hipStreamCreate(&hip_stream);
    hipGraphicsMapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsMapResources(1, &cuda_texture, hip_stream);

    size_t size_mapped;
    hipGraphicsSubResourceGetMappedArray(&array_ptr, cuda_texture, 0, 0);
    hipGraphicsResourceGetMappedPointer((void **)&mem_ptr, &size_mapped,
                                         cuda_buffer);
    assert(size_mapped == size_pixels * 4 * sizeof(float));  // RGBA32F

    // run kernel
    const int num_blocks = (size_pixels + BLOCK_SIZE - 1) / BLOCK_SIZE;
    cuda_render_test_kernel<<<num_blocks, BLOCK_SIZE>>>(w, h, mem_ptr);

    // unmap resources
    hipGraphicsUnmapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsUnmapResources(1, &cuda_texture, hip_stream);
    hipStreamDestroy(hip_stream);
}

void cuda_destroy() {}

__global__ void cuda_render_test_kernel(size_t w, size_t h, float *mem_ptr) {
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < w * h; i += stride) {
        const size_t idx = i * 4;
        const size_t x = i % w;
        const size_t y = i / w;
        float f = (Float3(x, y, 0) / Float3(w, h, 1) - Float3(0.5, 0.5, 0)).length();
        mem_ptr[idx + 0] = (float)x / w;
        mem_ptr[idx + 1] = (float)y / h;
        mem_ptr[idx + 2] = f;
        mem_ptr[idx + 3] = 1.f;
    }
}