#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <iostream>
#include <cuda_gl_interop.h>
#include "cuda_render.cuh"

void cuda_init(GLuint texture_id, GLuint buffer_id) {
    // register GL buffer and texture as CUDA resources
    hipGraphicsGLRegisterBuffer(&cuda_buffer, buffer_id, hipGraphicsRegisterFlagsNone);
    hipGraphicsGLRegisterImage(&cuda_texture, texture_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);

    // create CUDA stream
    hipStreamCreate(&hip_stream);
}

void cuda_render(GLuint buffer_id, size_t w, size_t h, const Mat4f& camera,
    std::vector<Geometry*> geom, unsigned iteration) {
    using namespace std;   
    
    size_t size;
    unsigned char *mem_ptr;
    hipArray *array_ptr;

    // map resources 
    hipGraphicsMapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsMapResources(1, &cuda_texture, hip_stream);
    
    hipGraphicsSubResourceGetMappedArray(&array_ptr, cuda_texture, 0, 0);
    hipGraphicsResourceGetMappedPointer((void **)&mem_ptr, &size, cuda_buffer);
    cuda_render_test_kernel<<<1, 1>>>();
    
    // unmap resources
    hipGraphicsUnmapResources(1, &cuda_buffer, hip_stream);
    hipGraphicsUnmapResources(1, &cuda_texture, hip_stream);
}

void cuda_destroy() {
    hipStreamDestroy(hip_stream);
}

__global__
void cuda_render_test_kernel() {

}